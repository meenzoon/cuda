#include <hip/hip_runtime.h>


extern "C"
__global__ void viterbi_kernel(
    int* observations, int* states, float* start_prob, float* trans_prob, float* emit_prob, int* path, float* delta, int* psi, int num_obs, int num_states) 
{
    int obs_idx = blockIdx.x;  // Observation index
    int state_idx = threadIdx.x;  // State index

    // Initialize delta and psi for the first observation
    if (obs_idx == 0) {
        delta[state_idx] = start_prob[state_idx] * emit_prob[state_idx * num_obs + observations[0]];
        psi[state_idx] = 0;
    } else {
        // For subsequent observations, calculate the maximum probability path
        float max_prob = -1.0f;
        int max_state = -1;

        for (int prev_state = 0; prev_state < num_states; ++prev_state) {
            float prob = delta[(obs_idx - 1) * num_states + prev_state] * trans_prob[prev_state * num_states + state_idx];
            if (prob > max_prob) {
                max_prob = prob;
                max_state = prev_state;
            }
        }

        delta[obs_idx * num_states + state_idx] = max_prob * emit_prob[state_idx * num_obs + observations[obs_idx]];
        psi[obs_idx * num_states + state_idx] = max_state;
    }

    // For the final step, find the most probable state
    if (obs_idx == num_obs - 1 && state_idx == 0) {
        float max_prob = -1.0f;
        int last_state = -1;

        for (int state = 0; state < num_states; ++state) {
            if (delta[(num_obs - 1) * num_states + state] > max_prob) {
                max_prob = delta[(num_obs - 1) * num_states + state];
                last_state = state;
            }
        }

        path[num_obs - 1] = last_state;

        // Backtrack to find the most probable path
        for (int t = num_obs - 2; t >= 0; --t) {
            path[t] = psi[(t + 1) * num_states + path[t + 1]];
        }
    }
}
